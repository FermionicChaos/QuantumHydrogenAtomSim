#include "hip/hip_runtime.h"
#include "CUDA_Functions_R2.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "stdud.h"
#include "VecR2.h"
using namespace std;

__global__ void cR2_SIN(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = sin(a[i]);
	}
}

__global__ void cR2_COS(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = cos(a[i]);
	}
}

__global__ void cR2_TAN(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		if (cos(a[i]) != 0.0) {
			b[i] = sin(a[i]) / cos(a[i]);
		}
		else {
			b[i] = 0.0;
		}
	}
}


__global__ void cR2_SINH(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = sinh(a[i]);
	}
}

__global__ void cR2_COSH(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = cosh(a[i]);
	}
}

__global__ void cR2_TANH(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = sinh(a[i]) / cosh(a[i]);
	}
}

__global__ void cR2_SQRT(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = sqrt(a[i]);
	}
}

__global__ void cR2_EXP(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = exp(a[i]);
	}
}

__global__ void cR2_POW(double* c, double* a, double b, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		c[i] = pow(a[i], b);
	}
}

__global__ void cR2_POW(double* c, double a, double* b, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		c[i] = pow(a, b[i]);
	}
}

__global__ void cR2_POW(double* c, double* a, double* b, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		c[i] = pow(a[i],b[i]);
	}
}

__global__ void cR2_ASIN(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		if (abs(a[i]) < 1.0) {
			b[i] = asin(a[i]);
		}
		else {
			b[i] = 0.0;
		}
	}
}

__global__ void cR2_ACOS(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		if (abs(a[i]) < 1.0) {
			b[i] = acos(a[i]);
		}
		else {
			b[i] = 0.0;
		}
	}
}

__global__ void cR2_ATAN(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = atan(a[i]);
	}
}

__global__ void cR2_ATAN(double* b, double* y, double* x, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = atan2(y[i], x[i]);
	}
}

__global__ void cR2_ERF(double* b, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = erf(a[i]);
	}
}

__global__ void cR2_J_n(double* b, int n, double* a, int N) {
	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {
		b[i] = jn(n, a[i]);
	}
}

cR2 sin(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_SIN << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 cos(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_COS << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 tan(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_TAN << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 sinh(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_SINH << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 cosh(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_COSH << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 tanh(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_TANH << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 sqrt(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_SQRT << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 exp(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_EXP << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 pow(const cR2 & base, double exp) {
	cR2 temp(base.getIndex1(), base.getIndex2());
	temp.setDX(base.getDX()); temp.setDY(base.getDY());
	temp.setX1(base.getX1()); temp.setX2(base.getX2());
	temp.setY1(base.getY1()); temp.setY2(base.getY2());
	temp.setG(base.getG()); temp.setB(base.getB());
	//Execute GPU duals:
	cR2_POW << <base.getG(), base.getB() >> > (temp.getD_ptr(), base.getD_ptr(), exp, base.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 pow(double base, const cR2 & exp) {
	cR2 temp(exp.getIndex1(), exp.getIndex2());
	temp.setDX(exp.getDX()); temp.setDY(exp.getDY());
	temp.setX1(exp.getX1()); temp.setX2(exp.getX2());
	temp.setY1(exp.getY1()); temp.setY2(exp.getY2());
	temp.setG(exp.getG()); temp.setB(exp.getB());
	//Execute GPU duals:
	cR2_POW << <exp.getG(), exp.getB() >> > (temp.getD_ptr(), base, exp.getD_ptr(), exp.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 pow(const cR2 & base, const cR2 & exp) {
	cR2 temp(base.getIndex1(), base.getIndex2());
	temp.setDX(base.getDX()); temp.setDY(base.getDY());
	temp.setX1(base.getX1()); temp.setX2(base.getX2());
	temp.setY1(base.getY1()); temp.setY2(base.getY2());
	temp.setG(base.getG()); temp.setB(base.getB());
	//Execute GPU duals:
	cR2_POW << <base.getG(), base.getB() >> > (temp.getD_ptr(), base.getD_ptr(), exp.getD_ptr(), base.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 asin(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_ASIN << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 acos(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_ACOS << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 atan(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_ATAN << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 atan(const cR2 & y, const cR2 & x) {
	cR2 temp(x.getIndex1(), x.getIndex2());
	temp.setDX(x.getDX()); temp.setDY(x.getDY());
	temp.setX1(x.getX1()); temp.setX2(x.getX2());
	temp.setY1(x.getY1()); temp.setY2(x.getY2());
	temp.setG(x.getG()); temp.setB(x.getB());
	//Execute GPU duals:
	cR2_ATAN << <x.getG(), x.getB() >> > (temp.getD_ptr(), y.getD_ptr(), x.getD_ptr(), x.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 erf(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_ERF << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 jn(int n, const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_J_n << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), n, arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}

cR2 ln(const cR2 & arg) {
	cR2 temp(arg.getIndex1(), arg.getIndex2());
	temp.setDX(arg.getDX()); temp.setDY(arg.getDY());
	temp.setX1(arg.getX1()); temp.setX2(arg.getX2());
	temp.setY1(arg.getY1()); temp.setY2(arg.getY2());
	temp.setG(arg.getG()); temp.setB(arg.getB());
	//Execute GPU duals:
	cR2_ERF << <arg.getG(), arg.getB() >> > (temp.getD_ptr(), arg.getD_ptr(), arg.getSize());

	//Optional: COPY to host, find better place!
	//hipMemcpy(h_yy, d_yy, m*n * sizeof(double), hipMemcpyDeviceToHost);
	return temp;
}
