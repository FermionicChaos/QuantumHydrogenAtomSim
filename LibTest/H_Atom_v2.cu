#include "hip/hip_runtime.h"
#include "H_Atom_v2.h"

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "stdud.h"
#include "extra.h"
using namespace std;
using namespace sf;

const double pi = 3.1415926535;

__global__ void Superposition(double* A, double* B, double* R, double* Theta,
	double* Psi, double* SVec,
	double tt, int I, int J, int K, int L) {

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = 0;
	A[i] = 0.0; B[i] = 0.0;
	for (int n = 1; n < L + 1; ++n) {
		for (int l = 0; l < n; ++l) {
			for (int m = -l; m < (l + 1); ++m) {
				A[i] += cos(tt/double(n*n))*SVec[k] * Psi[i + I*J*k];
				B[i] += -sin(tt/double(n*n))*SVec[k] * Psi[i + I*J*k];
				k += 1;
			}
		}
	}//*/
	R[i] = sqrt((A[i] * A[i]) + (B[i] * B[i]));
	Theta[i] = atan2(B[i], A[i]);
}
//Evaluate for each eigenfunction.
// C|nlm><nlm|f> = 
__global__ void EFP(double* lambda_R, double* lambda_Theta,
	double* r, double* theta, double* R, double* Theta) {

	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = 0;
	lambda_R[i] = r[i] * R[i];
	lambda_Theta[i] = theta[i] - Theta[i];
}

__global__ void Superposition2(double* A, double* B, double* R, double* Theta,
	double* Psi, double* AVec, double* BVec,
	double tt, int I, int J, int K, int L) {
	//Includes complex eigen vectors.
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int k = 0;
	A[i] = 0.0; B[i] = 0.0;
	for (int n = 1; n < L + 1; ++n) {
		for (int l = 0; l < n; ++l) {
			for (int m = -l; m < (l + 1); ++m) {
				A[i] += cos(2.0*tt / double(n*n))*AVec[k] * Psi[i + I*J*k];
				B[i] += -sin(2.0*tt / double(n*n))*BVec[k] * Psi[i + I*J*k];
				k += 1;
			}
		}
	}//*/
	R[i] = sqrt((A[i] * A[i]) + (B[i] * B[i]));
	Theta[i] = atan2(B[i], A[i]);
}

__global__ void add(double* b, double* a, int I) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j = blockDim.y * blockIdx.y + threadIdx.y;
	b[i + j*I] = a[i + j*I] * a[i + j*I];
}


H_Atom_v2::H_Atom_v2() {

}

H_Atom_v2::~H_Atom_v2() {
	//delete[] h_Ef;
	//delete[] h_vec;
	//delete[] h_pix;
	hipFree(d_Ef);
	hipFree(d_avec);
	hipFree(d_bvec);
	hipFree(d_pix);
	hipFree(d_psia);
	hipFree(d_psib);
	hipFree(d_R);
	hipFree(d_phase);
}

void H_Atom_v2::StateVec(double* psia, double* psib, cR2 & x, cR2 & y, int N) {
	tt = 0.0;
	h_avec = new double[N];
	h_bvec = new double[N];
	h_avec = psia; h_bvec = psib;
	dx = x.getDX(); dy = y.getDY();
	I = x.getIndex1(); J = y.getIndex2();
	Nh = N;
	K = N*(N + 1)*(2 * N + 1) / 6; //Develops energy eigenstate basis
	block = 1024;
	grid = I*J / 1024;
	alpha = x; phase = y;
	h_psia = new double[I*J];
	h_psib = new double[I*J];
	h_phase = new double[I*J];
	h_R = new double[I*J];
	hipMalloc(&d_psia, I*J * sizeof(double));
	hipMalloc(&d_psib, I*J * sizeof(double));
	hipMalloc(&d_phase, I*J * sizeof(double));
	hipMalloc(&d_R, I*J * sizeof(double));

	h_Ef = new double[I*J*K];
	hipMalloc((void **)&d_Ef, I*J*K * sizeof(double));

	cR2 temp;
	int index = 0;
	cout << "<x,y,z|n,l,m>" << endl;
	for (int n = 1; n < N + 1; ++n) {
		for (int l = 0; l < n; ++l) {
			for (int m = -l; m < (l + 1); ++m) {
				cout << "|";
				cout << n << "," << l << "," << m;
				cout << ">" << endl;

				temp = Psi_nlm(n, l, m, x, y);
				temp.call2Host();
				for (int i = 0; i < I*J; ++i) {
					h_Ef[i + I*J*index] = temp.getH_ptr()[i];
				}

				cout << "Index number:  " << index << endl;
				index += 1;
			}
		}
	}
	hipMemcpy(d_Ef, h_Ef, I*J*K * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_avec, h_avec, K * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_bvec, h_bvec, K * sizeof(double), hipMemcpyHostToDevice);
	cout << "Data Generated!" << endl;
}
/*
What we need, State vector from hilbert space E states |n,l,m>
|Psi> = SUM(C_k * |n,l,m>);
The eigenstate vector length l;
The dimension of the screen vertical 720 x horizontal 1280.
|Psi> = C_0*|1,0,0> + C_1*|2,0,0> + C_3*|2,1,-1> + ...
Position projection
<r|Psi> = C_0*<r|1,0,0> + C_1*<r|2,0,0> + C_3*<r|2,1,-1> + ...
Time evolution:
<r|Psi> = C_0*exp(-t)*<r|1,0,0> + C_1*exp(-t/4)*<r|2,0,0> + C_3*exp(-t/4)*<r|2,1,-1> + ...
Step 1: Generate eigen functions
Step 2: Time evolve for small dt
Step 3: Convert data into pixel data.
Step 4: Draw.

A point in space will be a complex number for all x,y in the set. The phase will determine the
color of the vertex, and the magnitude will determine the alpha opacity, after which the data is normalized
to create the pixel data.

gggg
*/
void H_Atom_v2::genData(int RESX,int RESY, cR2& x, cR2& y, int N) {

	tt = 0.0;
	I = RESX; J = RESY;
	dx = x.getDX(); dy = y.getDY();
	Nh = N;
	K = N*(N + 1)*(2 * N + 1) / 6; //Develops energy eigenstate basis
	block = 512;
	grid = I*J / 512;
	//Yeah, I know, it's a lot of threads. fite me
	alpha = x; phase = y;
	h_psia = new double[RESX*RESY];
	h_psib = new double[RESX*RESY];
	h_phase = new double[RESX*RESY];
	h_R = new double[RESX*RESY];
	hipMalloc(&d_psia, RESX*RESY * sizeof(double));
	hipMalloc(&d_psib, RESX*RESY * sizeof(double));
	hipMalloc(&d_phase, RESX*RESY * sizeof(double));
	hipMalloc(&d_R, RESX*RESY * sizeof(double));

	h_Ef = new double[RESX*RESY*K];
	hipMalloc((void **)&d_Ef, I*J*K * sizeof(double));
	h_avec = new double[K];
	hipMalloc((void **)&d_avec, K * sizeof(double));
	h_bvec = new double[K];
	hipMalloc((void **)&d_bvec, K * sizeof(double));

	cR2 temp;
	int index = 0;
	cout << "<x,y,z|n,l,m>" << endl;
	for (int n = 1; n < N + 1; ++n) {
		for (int l = 0; l < n; ++l) {
			for (int m = -l; m < (l + 1); ++m) {
				cout << "|";
				cout << n << "," << l << "," << m;
				cout << ">" << endl;

				temp = Psi_nlm(n, l, m, x, y);
				temp.call2Host();
				//cout << temp(0.0, 0.0) << endl;
				for (int i = 0; i < I*J; ++i) {
						h_Ef[i + I*J*index] = temp.getH_ptr()[i];
				}

				h_avec[index] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
				h_bvec[index] = static_cast <double> (rand()) / static_cast <double> (RAND_MAX);
				//h_avec[index] = 1.0;
				//h_bvec[index] = 0.0;
				cout << "Index number:  " << index << endl;
				index += 1;
			}
		}
	}
	//Update you fucking cock, nao
	hipMemcpy(d_Ef, h_Ef, I*J*K*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_avec, h_avec, K*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_bvec, h_bvec, K * sizeof(double), hipMemcpyHostToDevice);
	cout << "Data Generated!" << endl;
}

void H_Atom_v2::genIm(int H, int W) {
	Im = VertexArray(Points, H*W);
	h = H; w = W;
}

void H_Atom_v2::compute(float dt) {
	tt += double(dt);

	/*
	COCK << <grid, block>> > (d_psia, d_psib, d_R, d_phase, d_Ef, d_vec, tt, I, J, K, Nh);
	hipMemcpy(h_psia, d_psia, I*J * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_psib, d_psib, I*J * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_phase, d_phase, I*J * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_R, d_R, I*J * sizeof(double), hipMemcpyDeviceToHost);
	//*/

	Superposition2 << <grid, block >> > (d_psia, d_psib, alpha.getD_ptr(), phase.getD_ptr(), d_Ef, d_avec, d_bvec, tt, I, J, K, Nh);
	//cout << tt << endl;
	
	///*
	//hipMemcpy(h_psia, d_psia, I*J * sizeof(double), hipMemcpyDeviceToHost);
	//hipMemcpy(h_psib, d_psib, I*J * sizeof(double), hipMemcpyDeviceToHost);

	//Normalize arrays for conversion to pixel data.
	alpha.call2Host();
	phase.call2Host();
	alpha = normalize(alpha);
	alpha.call2Host();
	

	/*
	Uint8 r, g, b, a;
	double k = 255.0;
	for (int j = 0; j < J; ++j) {
		for (int i = 0; i < I; ++i) {
			r = 0; g = 0; b = 0; a = 0;
			if (cos(phase(i, j)) > 0.0) {
				r = static_cast<Uint8>(k*cos(phase(i, j)));
			}
			if (cos(phase(i, j) + 2 * pi / 3) > 0.0) {
				g = static_cast<Uint8>(k*cos(phase(i, j) + 2 * pi / 3));
			}
			if (cos(phase(i, j) - 2 * pi / 3) > 0.0) {
				b = static_cast<Uint8>(k*cos(phase(i, j) - 2 * pi / 3));
			}
			if (abs(alpha(i,j)) > 0.0) {
				a = static_cast<char>(k*alpha(i,j));
			}
			Im[i + j*w].position = Vector2f(i, j);
			Im[i + j*w].color = Color(r,g,b,a);
			Im[i + j*w].texCoords = Vector2f(i, j);
		}
	}//*/
	///*
	Uint8 r, g, b, a;
	double k = 255.0;
	for (int j = 0; j < J; ++j) {
		for (int i = 0; i < I; ++i) {
			r = 0; g = 0; b = 0; a = 0;
			r = static_cast<Uint8>(k*0.5*(cos(phase(i, j)) + 1));
			g = static_cast<Uint8>(k*0.5*(cos(phase(i, j) + 2 * pi / 3) + 1.0));
			b = static_cast<Uint8>(k*0.5*(cos(phase(i, j) - 2 * pi / 3) + 1.0));
			a = static_cast<char>(k*alpha(i, j));
			Im[i + j*w].position = Vector2f(i, j);
			Im[i + j*w].color = Color(r, g, b, a);
			Im[i + j*w].texCoords = Vector2f(i, j);
		}
	}
	//*/
}

void H_Atom_v2::draw(RenderWindow & window) {
	window.draw(Im);
}

void H_Atom_v2::ArrayTest() {
	double h_A[9];
	double *d_A;
	double h_B[9];
	double *d_B;
	hipMalloc(&d_A, 9*sizeof(double));
	hipMalloc(&d_B, 9*sizeof(double));
	for (int i = 0; i < 9; ++i) {
		h_A[i] = double(i);
		cout << h_A[i] << endl;
	}
	hipMemcpy(d_A, h_A, 9 * sizeof(double), hipMemcpyHostToDevice);
	dim3 g;
	g = dim3(3, 3);
	dim3 b;
	b = dim3(1, 1);
	add << <g, b >> > (d_B, d_A, 3);
	
	hipMemcpy(h_B, d_B, 9 * sizeof(double), hipMemcpyDeviceToHost);
	for (int j = 0; j < 9; ++j) {
		cout << h_A[j] << endl;
	}
	hipFree(d_A);
	hipFree(d_B);
}
