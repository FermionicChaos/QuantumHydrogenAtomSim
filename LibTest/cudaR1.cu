#include "hip/hip_runtime.h"
#include "cudaR1.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""

#include "stdud.h"


using namespace std;
/*
DEPENDENT ON CUDA 8.0!
*/

__global__ void cudaR1_Add(double* c, double* a, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void cudaR1_Sub(double* c, double* a, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] - b[i];
}

__global__ void cudaR1_Mult(double* c, double* a, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * b[i];
}

__global__ void cudaR1_Divide(double* c, double* a, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (b[i] != 0.0) {
		c[i] = a[i] / b[i];
	}
	else {
		c[i] = 0.0;
	}
}

__global__ void cudaR1_Add_RHS(double* c, double* a, double rhs) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + rhs;
}

__global__ void cudaR1_Sub_RHS(double* c, double* a, double rhs) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] - rhs;
}

__global__ void cudaR1_Mult_RHS(double* c, double* a, double rhs) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] * rhs;
}

__global__ void cudaR1_Divide_RHS(double* c, double* a, double rhs) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] / rhs;
}

__global__ void cudaR1_Add_LHS(double* c, double lhs, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = lhs + b[i];
}

__global__ void cudaR1_Sub_LHS(double* c, double lhs, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = lhs - b[i];
}

__global__ void cudaR1_Mult_LHS(double* c, double lhs, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = lhs * b[i];
}

__global__ void cudaR1_Divide_LHS(double* c, double lhs, double* b) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (b[i] != 0.0) {
		c[i] = lhs / b[i];
	}
	else {
		c[i] = 0.0;
	}
}

cudaR1::cudaR1() {
	ptr = nullptr;
	x1 = 0; x2 = 0, dx = 0; m = 0;
	grid = 0; block = 0;
}

cudaR1::~cudaR1() {
	//Destructor
	//cout << "MEMORY De-Allocated: ADDRESS -> " << ptr << endl;
	delete[] ptr;
}

cudaR1::cudaR1(const cudaR1& rhs) {
	//Copy Constructor SEPERATE POINTER VALUES FOR DIFFERENT OBJECTS
	//cout << "Copy Constructor" << endl;
	m = rhs.getSize(); dx = rhs.getDX();
	x1 = rhs.getX1(); x2 = rhs.getX2();
	grid = rhs.grid; block = rhs.block;
	double* BR = rhs.ptr;
	ptr = new double[m];
	//cout << "COPIED From: ADDRESS -> " << rhs.getptr() << endl;
	//cout << "COPIED To: -> " << ptr << endl;
	for (int i = 0; i < m; ++i) {
		ptr[i] = BR[i];
	}
	//cout << "END: COPY Constructor Complete" << endl;
}

cudaR1& cudaR1::operator=(const cudaR1& rhs) {
	//cout << "Operation: ASSIGNMENT" << endl;
	//cout << "LEFT ADDRESS -> " << ptr << endl;
	//cout << "RIGHT ADDRESS -> " << rhs.getptr() << endl;
	//cout << rhs << endl;
	if ((ptr != rhs.getBase())) {
		this->m = rhs.getSize(); this->dx = rhs.getDX();
		this->x1 = rhs.getX1(); this->x2 = rhs.getX2();
		this->grid = rhs.grid; this->block = rhs.block;
		delete[] this->ptr;
		ptr = new double[m];
		for (int i = 0; i < m; ++i) {
			ptr[i] = rhs.getBase()[i];
		}
		this->ptr = ptr;
		return *this;
	}
	else {
		return *this;
	}
}

cudaR1::cudaR1(cudaR1&& inp) {
	ptr = inp.ptr;
	inp.ptr = nullptr;
	m = inp.m; dx = inp.dx;
	x1 = inp.x1; x2 = inp.x2;
	grid = inp.grid; block = inp.block;
}

cudaR1& cudaR1::operator=(cudaR1&& rhs) {
	delete[] ptr;
	ptr = rhs.ptr;
	m = rhs.m; dx = rhs.dx;
	x1 = rhs.x1; x2 = rhs.x2;
	grid = rhs.grid; block = rhs.block;
	rhs.ptr = nullptr;
	return *this;
}

cudaR1::cudaR1(int size) {
	m = size; x1 = 0; x2 = 0; dx = 0;
	ptr = new double[m];
}

cudaR1::cudaR1(double a, double b, int res) {
	//Large latency, initiate for domain.
	//Initiate as soon as possible for other operations.
	if ((res%256) == 0) {
		x1 = a; x2 = b; m = res;
		grid = (m / 256); block = 256;
		dx = (b - a) / (double(res));
		ptr = new double[res];
		for (int i = 0; i < m; ++i) {
			ptr[i] = x1 + double(i)*dx;
		}
		x2 = ptr[m - 1];
	}
	else {
		cout << "Must be a multiple of 256!" << endl;
	}
}

double cudaR1::operator()(double p0) {
	double val, p;
	int i;
	p = (p0 - x1) / dx;
	i = int(p);
	if ((i > 0) && (i < m - 1)) {
		val = ptr[i];
		return val;
	}
	else {
		return val = 0;
	}
	return val;
}

double cudaR1::operator()(int i) {
	double val;
	if ((i > 0) && (i < m - 1)) {
		val = ptr[i];
		return val;
	}
	else {
		return val = 0;
	}
}

cudaR1 cudaR1::operator+(const cudaR1 & rhs) {
	cudaR1 temp;
	if ((x1 = rhs.x1) && (x2 = rhs.x2) && (m = rhs.m) && (dx = rhs.dx)) {
		double* h_xx1 = ptr;
		double* h_xx2 = rhs.getBase();
		double* h_yy = new double[rhs.getSize()];
		temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
		temp.setX1(x1); temp.setX2(x2);
		//GPU duals.
		double *d_xx1, *d_xx2, *d_yy;
		hipError_t cudaStatus;
		//Check:
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		//Reserve:
		cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_xx2, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_yy, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		//Copy2d:
		cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// grid has blocks, and a block has multi threads.
		//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
		cudaR1_Add<<<grid, block>>>(d_yy, d_xx1, d_xx2);

		//Sync:	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		//Return2h:
		cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		//Transfer back:
		//temp.setBase(h_yy);
		//ReleaseMEM:
		hipFree(d_xx1); hipFree(d_xx2); hipFree(d_yy);
		cout << "Add Success!: " << false << endl;
		return temp;

	Error:
		hipFree(d_xx1);
		hipFree(d_xx2);
		hipFree(d_yy);
		cout << "Error!" << true << endl;
		return temp;
	}
	else {
		cout << "Error: Dimensions must agree!" << endl;
		return temp;
	}
}

cudaR1 cudaR1::operator-(const cudaR1 & rhs) {
	cudaR1 temp;
	if ((x1 = rhs.x1) && (x2 = rhs.x2) && (m = rhs.m) && (dx = rhs.dx)) {
		double* h_xx1 = ptr;
		double* h_xx2 = rhs.getBase();
		double* h_yy = new double[rhs.getSize()];
		temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
		temp.setX1(x1); temp.setX2(x2);
		//GPU duals.
		double *d_xx1, *d_xx2, *d_yy;
		hipError_t cudaStatus;
		//Check:
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		//Reserve:
		cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_xx2, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_yy, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		//Copy2d:
		cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// grid has blocks, and a block has multi threads.
		//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
		cudaR1_Sub<<<grid, block >>>(d_yy, d_xx1, d_xx2);

		//Sync:	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		//Return2h:
		cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		//Transfer back:
		//temp.setBase(h_yy);
		//ReleaseMEM:
		hipFree(d_xx1); hipFree(d_xx2); hipFree(d_yy);
		cout << "Add Success!: " << false << endl;
		return temp;

	Error:
		hipFree(d_xx1);
		hipFree(d_xx2);
		hipFree(d_yy);
		cout << "Error!" << true << endl;
		return temp;
	}
	else {
		cout << "Error: Dimensions must agree!" << endl;
		return temp;
	}
}

cudaR1 cudaR1::operator*(const cudaR1 & rhs) {
	cudaR1 temp;
	if ((x1 = rhs.x1) && (x2 = rhs.x2) && (m = rhs.m) && (dx = rhs.dx)) {
		double* h_xx1 = ptr;
		double* h_xx2 = rhs.getBase();
		double* h_yy = new double[rhs.getSize()];
		temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
		temp.setX1(x1); temp.setX2(x2);
		//GPU duals.
		double *d_xx1, *d_xx2, *d_yy;
		hipError_t cudaStatus;
		//Check:
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		//Reserve:
		cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_xx2, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_yy, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		//Copy2d:
		cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// grid has blocks, and a block has multi threads.
		//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
		cudaR1_Mult<<<grid, block >>>(d_yy, d_xx1, d_xx2);

		//Sync:	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		//Return2h:
		cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		//Transfer back:
		//temp.setBase(h_yy);
		//ReleaseMEM:
		hipFree(d_xx1); hipFree(d_xx2); hipFree(d_yy);
		cout << "Add Success!: " << false << endl;
		return temp;

	Error:
		hipFree(d_xx1);
		hipFree(d_xx2);
		hipFree(d_yy);
		cout << "Error!" << true << endl;
		return temp;
	}
	else {
		cout << "Error: Dimensions must agree!" << endl;
		return temp;
	}
}

cudaR1 cudaR1::operator/(const cudaR1 & rhs) {
	cudaR1 temp;
	if ((x1 = rhs.x1) && (x2 = rhs.x2) && (m = rhs.m) && (dx = rhs.dx)) {
		double* h_xx1 = ptr;
		double* h_xx2 = rhs.getBase();
		double* h_yy = new double[rhs.getSize()];
		temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
		temp.setX1(x1); temp.setX2(x2);
		//GPU duals.
		double *d_xx1, *d_xx2, *d_yy;
		hipError_t cudaStatus;
		//Check:
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		//Reserve:
		cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_xx2, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_yy, rhs.getSize() * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		//Copy2d:
		cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// grid has blocks, and a block has multi threads.
		//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
		cudaR1_Divide<<<grid, block >>>(d_yy, d_xx1, d_xx2);

		//Sync:	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		//Return2h:
		cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		//Transfer back:
		//temp.setBase(h_yy);
		//ReleaseMEM:
		hipFree(d_xx1); hipFree(d_xx2); hipFree(d_yy);
		cout << "Add Success!: " << false << endl;
		return temp;

	Error:
		hipFree(d_xx1);
		hipFree(d_xx2);
		hipFree(d_yy);
		cout << "Error!" << true << endl;
		return temp;
	}
	else {
		cout << "Error: Dimensions must agree!" << endl;
		return temp;
	}
}

cudaR1 cudaR1::operator+(double rhs) {
	cudaR1 temp;
	double* h_xx1 = ptr;
	//double* hRHS = &rhs;
	double* h_yy = new double[m];
	temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
	temp.setX1(x1); temp.setX2(x2);
	//GPU duals.
	double *d_xx1, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	cudaStatus = hipMalloc(&d_xx1, m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	/*cudaStatus = hipMalloc(&d_rhs, sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}*/
	cudaStatus = hipMalloc(&d_yy, m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	cudaStatus = hipMemcpy(d_xx1, h_xx1, m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	/*cudaStatus = hipMemcpy(d_rhs, &rhs, sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Add_RHS<<<grid, block>>>(d_yy, d_xx1, rhs);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, m * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx1); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	hipFree(d_xx1);
	//hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
	return cudaR1();
}

cudaR1 cudaR1::operator-(double rhs) {
	cudaR1 temp;
	double* h_xx1 = ptr;
	//double* hRHS = &rhs;
	double* h_yy = new double[m];
	temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
	temp.setX1(x1); temp.setX2(x2);
	//GPU duals.
	double *d_xx1, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	cudaStatus = hipMalloc(&d_xx1, m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	/*cudaStatus = hipMalloc(&d_rhs, sizeof(double));
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/
	cudaStatus = hipMalloc(&d_yy, m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	cudaStatus = hipMemcpy(d_xx1, h_xx1, m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	/*cudaStatus = hipMemcpy(d_rhs, &rhs, sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Sub_RHS<<<grid, block >>>(d_yy, d_xx1, rhs);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, m * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx1); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	hipFree(d_xx1);
	//hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
	return cudaR1();
}

cudaR1 cudaR1::operator*(double rhs) {
	cudaR1 temp;
	double* h_xx1 = ptr;
	//double* hRHS = &rhs;
	double* h_yy = new double[m];
	temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
	temp.setX1(x1); temp.setX2(x2);
	//GPU duals.
	double *d_xx1, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	cudaStatus = hipMalloc(&d_xx1, m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	/*cudaStatus = hipMalloc(&d_rhs, sizeof(double));
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/
	cudaStatus = hipMalloc(&d_yy, m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	cudaStatus = hipMemcpy(d_xx1, h_xx1, m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	/*cudaStatus = hipMemcpy(d_rhs, &rhs, sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Mult_RHS<<<grid, block>>>(d_yy, d_xx1, rhs);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, m * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx1); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	hipFree(d_xx1);
	//hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
	return cudaR1();
}

cudaR1 cudaR1::operator/(double rhs) {
	cudaR1 temp;
	if (rhs != 0.0) {
		double* h_xx1 = ptr;
		//double* hRHS = &rhs;
		double* h_yy = new double[m];
		temp.setBase(h_yy); temp.setDX(dx); temp.setSize(m);
		temp.setX1(x1); temp.setX2(x2);
		//GPU duals.
		double *d_xx1, *d_yy;
		hipError_t cudaStatus;
		//Check:
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		//Reserve:
		cudaStatus = hipMalloc(&d_xx1, m * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		/*cudaStatus = hipMalloc(&d_rhs, sizeof(double));
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
		}*/
		cudaStatus = hipMalloc(&d_yy, m * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		//Copy2d:
		cudaStatus = hipMemcpy(d_xx1, h_xx1, m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		/*cudaStatus = hipMemcpy(d_rhs, &rhs, sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
		}*/

		// grid has blocks, and a block has multi threads.
		//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
		cudaR1_Divide_RHS<<<grid, block>>>(d_yy, d_xx1, rhs);

		//Sync:	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		//Return2h:
		cudaStatus = hipMemcpy(h_yy, d_yy, m * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		//Transfer back:
		//temp.setBase(h_yy);
		//ReleaseMEM:
		hipFree(d_xx1); hipFree(d_yy);
		cout << "Add Success!: " << false << endl;
		return temp;

	Error:
		hipFree(d_xx1);
		//hipFree(d_xx2);
		hipFree(d_yy);
		cout << "Error!" << true << endl;
		return temp;
	}
	else {
		cout << "Error: Division by zero is not allowed!" << endl;
		return temp;
	}
}

void cudaR1::InitGB() {
	//Initialize grid/block incase forgotten.
	block = 256;
	grid = m/256;
};

cudaR1 operator+(double lhs, const cudaR1& rhs) {
	cudaR1 temp;
	//double* h_xx1 = ptr;
	double* h_xx2 = rhs.ptr;
	double* h_yy = new double[rhs.m];
	temp.setBase(h_yy); temp.setDX(rhs.dx); temp.setSize(rhs.m);
	temp.setX1(rhs.x1); temp.setX2(rhs.x2);
	//GPU duals.
	double *d_xx2, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	/*cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}*/
	cudaStatus = hipMalloc(&d_xx2, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc(&d_yy, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	/*cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}*/
	cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Add_LHS<<<rhs.grid, rhs.block>>>(d_yy, lhs, d_xx2);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx2); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	//hipFree(d_xx1);
	hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
}

cudaR1 operator-(double lhs, const cudaR1 & rhs) {
	cudaR1 temp;
	//double* h_xx1 = ptr;
	double* h_xx2 = rhs.ptr;
	double* h_yy = new double[rhs.m];
	temp.setBase(h_yy); temp.setDX(rhs.dx); temp.setSize(rhs.m);
	temp.setX1(rhs.x1); temp.setX2(rhs.x2);
	//GPU duals.
	double *d_xx2, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	/*cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/
	cudaStatus = hipMalloc(&d_xx2, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc(&d_yy, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	/*cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/
	cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Sub_LHS<<<rhs.grid, rhs.block>>>(d_yy, lhs, d_xx2);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx2); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	//hipFree(d_xx1);
	hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
}

cudaR1 operator*(double lhs, const cudaR1 & rhs) {
	cudaR1 temp;
	//double* h_xx1 = ptr;
	double* h_xx2 = rhs.ptr;
	double* h_yy = new double[rhs.m];
	temp.setBase(h_yy); temp.setDX(rhs.dx); temp.setSize(rhs.m);
	temp.setX1(rhs.x1); temp.setX2(rhs.x2);
	//GPU duals.
	double *d_xx2, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	/*cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/
	cudaStatus = hipMalloc(&d_xx2, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc(&d_yy, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	/*cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/
	cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Mult_LHS<<<rhs.grid, rhs.block>>>(d_yy, lhs, d_xx2);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx2); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	//hipFree(d_xx1);
	hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
}

cudaR1 operator/(double lhs, const cudaR1 & rhs) {
	cudaR1 temp;
	//double* h_xx1 = ptr;
	double* h_xx2 = rhs.ptr;
	double* h_yy = new double[rhs.m];
	temp.setBase(h_yy); temp.setDX(rhs.dx); temp.setSize(rhs.m);
	temp.setX1(rhs.x1); temp.setX2(rhs.x2);
	//GPU duals.
	double *d_xx2, *d_yy;
	hipError_t cudaStatus;
	//Check:
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	//Reserve:
	/*cudaStatus = hipMalloc(&d_xx1, rhs.getSize() * sizeof(double));
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMalloc failed!");
	goto Error;
	}*/
	cudaStatus = hipMalloc(&d_xx2, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc(&d_yy, rhs.m * sizeof(double));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	//Copy2d:
	/*cudaStatus = hipMemcpy(d_xx1, h_xx1, rhs.getSize() * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
	goto Error;
	}*/
	cudaStatus = hipMemcpy(d_xx2, h_xx2, rhs.m * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// grid has blocks, and a block has multi threads.
	//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
	cudaR1_Divide_LHS << <rhs.grid, rhs.block >> >(d_yy, lhs, d_xx2);

	//Sync:	
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}
	//Return2h:
	cudaStatus = hipMemcpy(h_yy, d_yy, rhs.getSize() * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	//Transfer back:
	//temp.setBase(h_yy);
	//ReleaseMEM:
	hipFree(d_xx2); hipFree(d_yy);
	cout << "Add Success!: " << false << endl;
	return temp;

Error:
	//hipFree(d_xx1);
	hipFree(d_xx2);
	hipFree(d_yy);
	cout << "Error!" << true << endl;
	return temp;
}

/*
cudaR1 d_dx(const cudaR1 & f, int ver) {
	cudaR1 temp;
	if (ver == 1) {
		double* h_ff = f.ptr;
		//double* h_xx2 = rhs.getBase();
		double* h_yy = new double[f.m];
		temp.setBase(h_yy); temp.setDX(f.dx); temp.setSize(f.m);
		temp.setX1(f.x1); temp.setX2(f.x2);
		//GPU duals.
		double *d_ff, *d_yy;
		hipError_t cudaStatus;
		//Check:
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		//Reserve:
		cudaStatus = hipMalloc(&d_ff, f.m* sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		cudaStatus = hipMalloc(&d_yy, f.m * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed!");
			goto Error;
		}
		//Copy2d:
		cudaStatus = hipMemcpy(d_ff, h_ff, f.m * sizeof(double), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}

		// grid has blocks, and a block has multi threads.
		//Execute: F : R1 -> R1 { xx1 + xx2 = yy }
		cudaR1_d_dx_Ver1<<<grid, block>>>(d_yy, d_ff, f.dx, );

		//Sync:	
		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Add launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto Error;
		}

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
			goto Error;
		}
		//Return2h:
		cudaStatus = hipMemcpy(h_yy, d_yy, f.m * sizeof(double), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto Error;
		}
		//Transfer back:
		//temp.setBase(h_yy);
		//ReleaseMEM:
		hipFree(d_ff); hipFree(d_yy);
		cout << "Add Success!: " << false << endl;
		return temp;

	Error:
		hipFree(d_ff);
		hipFree(d_yy);
		cout << "Error!" << true << endl;
		return temp;
	}
	else if (ver == 2) {

	}
	else if (ver == 3) {

	}
	else {
		cout << "Invalid!" << endl;
	}
}*/
